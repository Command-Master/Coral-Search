
#include <hip/hip_runtime.h>
#include <stdio.h>
#define ll long long int

const ll m = 0x5DEECE66Dll;
const ll mask = (1ll << 48) - 1;

#define advance1(s) s = (s * m + 11ll) & mask
#define advance2(s)  \
    advance1(s); \
    advance1(s);
#define getIntBounded(x, s, n)              \
    if ((n & (-n)) == n)                    \
    {                                       \
        advance1(s);                        \
        x = (int)((n * (s >> 17)) >> 31);   \
    }                                       \
    else                                    \
    {                                       \
        do                                  \
        {                                   \
            advance1(s);                    \
            bits = s >> 17;                 \
            val = bits % n;                 \
        } while (bits - val + (n - 1) < 0); \
        x = val;                            \
    }
#define nextFloat(x, s) \
    x = ((advance1(s)) >> 32) / 1.6777216E7F;
#define swap(x, y) \
    x += y;        \
    y = x;         \
    x -= y;

__device__
void check_seed(ll seed)
{
    ll color, stem_height;
    ll bits, val;
    float temp;
    ll tempi;
    getIntBounded(color, seed, 5);
    if (color != 0)
    {
        return;
    }
    getIntBounded(stem_height, seed, 5);
    if (stem_height != 0)
    {
        return;
    }
    nextFloat(temp, seed);
    if (temp < 0.25f)
    {
        return;
    }
    nextFloat(temp, seed);
    if (temp < 0.05f)
    {
        return;
    }

    nextFloat(temp, seed);
    if (temp < 0.2f)
    {
        return;
    }

    nextFloat(temp, seed);
    if (temp < 0.2f)
    {
        return;
    }

    nextFloat(temp, seed);
    if (temp < 0.2f)
    {
        return;
    }

    nextFloat(temp, seed);
    if (temp < 0.2f)
        getIntBounded(tempi, seed, 5); // unknown, dynamic skip

    // 2 sides:
    getIntBounded(tempi, seed, 3);
    if (tempi != 0)
    {
        return;
    }

    int a[] = {0, 1, 2, 3};
    /*
    4, 3, 2 as x:
     swap(a[x-1], a[nextInt(x)])
    */
    getIntBounded(tempi, seed, 4);
    swap(a[3], a[tempi]);
    getIntBounded(tempi, seed, 3);
    swap(a[2], a[tempi]);
    getIntBounded(tempi, seed, 2);
    swap(a[1], a[tempi]);
    // 2 = south, 3 = west
    if (a[0] + a[1] != 5)
    {
        return;
    }
    int i;
    for (i = 0; i < 2; i++)
    {
        // height
        getIntBounded(tempi, seed, 5);
        if (tempi != 1)
            return;

        if (a[i] == 2)
        {
            nextFloat(temp, seed);
            if (temp < 0.25f)
                return;
            nextFloat(temp, seed);
            if (temp < 0.05f)
                return;
            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
                getIntBounded(tempi, seed, 5); // unknown, dynamic skip

            advance2(seed); // skip sea pickles and corals which would get overriten

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp >= 0.2f)
            {
                return;
            }

            getIntBounded(tempi, seed, 5);
            if (tempi != 3)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
                getIntBounded(tempi, seed, 5); // unknown, dynamic skip

            // coral on top:
            nextFloat(temp, seed);
            if (temp >= 0.25f)
            {
                return;
            }
            getIntBounded(tempi, seed, 5);
            if (tempi != 4) {
                return;
            }
            

            // wall corals:
            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp >= 0.2f)
            {
                return;
            }
            getIntBounded(tempi, seed, 5);
            if (tempi != 4)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }


            nextFloat(temp, seed);
            if (temp < 0.2f)
                getIntBounded(tempi, seed, 5); // unknown, dynamic skip
        }
        else if (a[i] == 3)
        {
            nextFloat(temp, seed);
            if (temp < 0.25f)
                return;
            nextFloat(temp, seed);
            if (temp < 0.05f)
                return;
            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp >= 0.2f)
            {
                return;
            }

            getIntBounded(tempi, seed, 5);
            if (tempi != 1)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2) getIntBounded(tempi, seed, 5); // unknown, dynamic skip

            advance2(seed); // skip sea pickles and corals which would get overriten

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
                getIntBounded(tempi, seed, 5); // unknown, dynamic skip

            // coral on top:
            nextFloat(temp, seed);
            if (temp >= 0.25f)
            {
                return;
            }
            getIntBounded(tempi, seed, 5);
            if (tempi != 1) {
                return;
            }
            

            // wall corals:
            nextFloat(temp, seed);
            if (temp >= 0.2f)
            {
                return;
            }
            getIntBounded(tempi, seed, 5);
            if (tempi != 0)
            {
                return;
            }


            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }

            nextFloat(temp, seed);
            if (temp < 0.2f)
            {
                return;
            }



            nextFloat(temp, seed);
            if (temp < 0.2f)
                getIntBounded(tempi, seed, 5); // unknown, dynamic skip
        }
    }

    printf("%lld\n", seed);
}

__global__
void find() {
    ll i = 0;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (i = index; i < (1ll << 48); i += stride) {
        check_seed(i);
    }
}

int main(void) {
    printf("Begin\n");
    int blockSize = 256;
    ll N = 1ll << 48;
    int numBlocks = (N + blockSize - 1) / blockSize;
    find<<<numBlocks, blockSize>>>();
    hipDeviceSynchronize();
    printf("End\n");
}
